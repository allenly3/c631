#include "hip/hip_runtime.h"
/*
    Compile on graham with:

    nvcc -arch=sm_60 -O2 q1p3.cu 

    nvprof ./a.out

    if want to see cpu profiling to compare GPU and CPU performance

    nvprof  --cpu-profiling on   ./a.out

*/
#include <stdio.h>
#include <math.h>
#include <float.h>

#define N 1024*1024  // 100 particles
#define BLOCK_SIZE 256 // Block size

__global__ void findMinimumDistance(double *particles, double *minDistance) {
    __shared__ double sharedMinDistances[BLOCK_SIZE];

    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    double myMinDistance = DBL_MAX;

    // Calculate the indices of the particles for this thread
    int particle1_index = tid * 2;
    int particle2_index = particle1_index + 2;

    // Ensure the indices are within bounds
    if (particle1_index < N * 2 && particle2_index < N * 2) {
        // Compute the distance between the particles
        double dx = particles[particle1_index] - particles[particle2_index];
        double dy = particles[particle1_index + 1] - particles[particle2_index + 1];
        double distance = sqrt(dx * dx + dy * dy);
        myMinDistance = distance;
    }

    sharedMinDistances[threadIdx.x] = myMinDistance;
    __syncthreads();

    // Reduction to find minimum distance among threads in the block
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            sharedMinDistances[threadIdx.x] = fmin(sharedMinDistances[threadIdx.x], sharedMinDistances[threadIdx.x + s]);
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        minDistance[blockIdx.x] = sharedMinDistances[0];
    }
}

int main() {
    double *particles_dev, *minDistance_dev;
    double particles_host[N * 2];
    double minDistance_host[N / BLOCK_SIZE + 1];

    srand(time(NULL));
    for (int i = 0; i < N * 2; ++i) {
        particles_host[i] = (double)rand() / RAND_MAX;
    }

    hipMalloc((void **)&particles_dev, N * 2 * sizeof(double));
    hipMalloc((void **)&minDistance_dev, (N / BLOCK_SIZE + 1) * sizeof(double));

    hipMemcpy(particles_dev, particles_host, N * 2 * sizeof(double), hipMemcpyHostToDevice);

    int numBlocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    findMinimumDistance<<<numBlocks, BLOCK_SIZE>>>(particles_dev, minDistance_dev);

    hipMemcpy(minDistance_host, minDistance_dev, (N / BLOCK_SIZE + 1) * sizeof(double), hipMemcpyDeviceToHost);

    double minDistance = minDistance_host[0];
    for (int i = 1; i < (N / BLOCK_SIZE + 1); ++i) {
        minDistance = fmin(minDistance, minDistance_host[i]);
    }

    printf("Minimum distance between particles (GPU): %lf\n", minDistance);

    hipFree(particles_dev);
    hipFree(minDistance_dev);

    return 0;
}
